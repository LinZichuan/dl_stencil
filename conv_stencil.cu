#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hipDNN.h>
#include <hipblas.h>
using namespace std;
#define real float
#define N 3
#define C 2
#define H 5
#define W 5
#define R 3
#define S 3
#define BX 1
#define BY 1
#define OH (H-R+1)
#define OW (W-S+1)
#define checkCUDNNError(status) \
    if (status != HIPDNN_STATUS_SUCCESS) { \
        printf("CUDA FAILURE: %s\n", hipdnnGetErrorString(status)); \
    } 
#define checkCudaError(status) \
    if (status != hipSuccess) { \
        printf("CUDA FAILURE: %s\n", hipGetErrorString(status)); \
    } 

float alpha=1.f, beta=0.f;
hipdnnHandle_t cudnnHandle;
hipdnnTensorDescriptor_t bottom_desc_, top_desc_; 
hipdnnFilterDescriptor_t filter_desc_;
hipdnnConvolutionDescriptor_t conv_desc_;
hipdnnConvolutionFwdAlgo_t algo_;
size_t workspaceSizeInBytes;
void* workspace;

void setup() {
    checkCUDNNError(hipdnnCreateTensorDescriptor(&bottom_desc_));
    checkCUDNNError(hipdnnCreateTensorDescriptor(&top_desc_));
    checkCUDNNError(hipdnnCreateFilterDescriptor(&filter_desc_));
    checkCUDNNError(hipdnnCreateConvolutionDescriptor(&conv_desc_));
    checkCUDNNError(hipdnnCreate(&cudnnHandle));
    checkCUDNNError(hipdnnSetTensor4dDescriptor(bottom_desc_,
                    HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
                    N, C, H, W));
    checkCUDNNError(hipdnnSetTensor4dDescriptor(top_desc_,
                    HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
                    N, 1, OH, OW)); //如果没有多个神经元，此处的output feature maps 应该等于1 ！！！否则cudnn会输出多个channel而导致结果错误！！
    checkCUDNNError(hipdnnSetFilter4dDescriptor(filter_desc_,
                    HIPDNN_DATA_FLOAT, 1, C, R, S)); //k,c,h,w  k=output_feature_maps; c=input_feature_maps
    checkCUDNNError(hipdnnSetConvolution2dDescriptor(conv_desc_,
                    0, 0, 1, 1, 1, 1, HIPDNN_CONVOLUTION)); //pad_h, pad_w, u(vertical stride), v(horizonal stride), upscalex=1, upscaley=1

    checkCUDNNError(hipdnnGetConvolutionForwardAlgorithm(cudnnHandle,
                    bottom_desc_, filter_desc_, conv_desc_, top_desc_,
                    HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &algo_));
    checkCUDNNError(hipdnnGetConvolutionForwardWorkspaceSize(cudnnHandle,
                    bottom_desc_, filter_desc_, conv_desc_, top_desc_,
                    algo_, &workspaceSizeInBytes));
    checkCudaError(hipMalloc((void**)&workspace, workspaceSizeInBytes));
}

__global__ void baseline(real* input, real* output, real* K, int outh, int outw) {
    int i = threadIdx.x + blockDim.x*blockIdx.x;
    int j = threadIdx.y + blockDim.y*blockIdx.y;
    float tmp;
    //注意边界thread的判断！！！
    if (i < W-S+1 && j < H-R+1) { 
        for (int n = 0; n < N; ++n) {
            tmp = 0;
            for (int c = 0; c < C; ++c) {
                for (int a = 0; a < R; ++a) {
                    for (int b = 0; b < S; ++b) {
                        tmp += input[((j+a)*W+(i+b)) + n*H*W*C + c*H*W] * K[(R-1-a)*S + (S-1-b) + c*R*S];
                    }
                }
            }
            output[j*outw+i + n*OH*OW] = tmp;
        }
    }
}

//fixed on 3*3
__global__ void opt_register(real* input, real* output, real* K, int outh, int outw) {
    int i = threadIdx.x + blockDim.x*blockIdx.x;
    int j = threadIdx.y + blockDim.y*blockIdx.y;
    /*int thi = threadIdx.x;*/
    int thj = threadIdx.y;
    float a0,a1,a2,a3,a4,a5,a6,a7,a8;
    if (thj == 0) {
        a0 = K[0];
        a1 = K[1];
        a2 = K[2];
        a3 = K[3];
        a4 = K[4];
        a5 = K[5];
        a6 = K[6];
        a7 = K[7];
        a8 = K[8];
    }
    __syncthreads();
    output[j*outw+i] = input[j*W+i]*a8 + input[j*W+i+1]*a7 + input[j*W+i+2]*a6 + 
                       input[(j+1)*W+i]*a5 + input[(j+1)*W+i+1]*a4 + input[(j+1)*W+i+2]*a3 +
                       input[(j+2)*W+i]*a2 + input[(j+2)*W+i+1]*a1 + input[(j+2)*W+i+2]*a0;
    /*output[i*outw+j] = input[j*W+i]*K[8] + input[j*W+i+1]*K[7] + input[j*W+i+2]*K[6] + */
                       /*input[(j+1)*W+i]*K[5] + input[(j+1)*W+i+1]*K[4] + input[(j+1)*W+i+2]*K[3] +*/
                       /*input[(j+2)*W+i]*K[2] + input[(j+2)*W+i+1]*K[1] + input[(j+2)*W+i+2]*K[0];*/
}
__global__ void opt_shm(real* input, real* output, real* K, int outh, int outw) {
    int i = threadIdx.x + blockDim.x*blockIdx.x;
    int j = threadIdx.y + blockDim.y*blockIdx.y;
    output[i*outw+j] = 0;
    __shared__ real shm_b[BY][BX];
    int thi = threadIdx.x;
    int thj = threadIdx.y;
    if (thi == 0 && thj == 0) {
        for (int a = 0; a < R; ++a) {
            for (int b = 0; b < S; ++b) {
                shm_b[a][b] = input[i*S+j];
            }
        }
    }
    for (int a = i; a < i+S; ++a) {
        for (int b = j; b < j+R; ++b) {
            output[i*outw+j] += shm_b[a][b] * K[(R-1-(b-j))*S+(S-1-(a-i))];
        }
    }
}
void cpu_comp(real* input, real* output, real* K, int h, int w, int kr, int ks, int outh, int outw) {
    for (int n = 0; n < N; ++n) {
        for (int i = 0; i < outh; ++i) {
            for (int j = 0; j < outw; ++j) {
                real a = 0;
                for (int c = 0; c < C; ++c) {
                    for (int r = 0; r < kr; ++r) {
                        for (int s = 0; s < ks; ++s) {
                            //a += input[(i+r)*w + (j+s)] * K[(kr-1-r)*ks+(ks-1-s)];
                            a += input[((i+r)*w+(j+s)) + n*H*W*C + c*H*W] * K[(kr-1-r)*ks+(ks-1-s) + c*R*S];
                        }
                    }
                }
                //output[(i*outw+j) + n*OH*OW*C + c*OH*OW] = a;
                output[(i*outw+j) + n*OH*OW] = a;
            }
        }
    }
}
bool check(real* A, real* B, int size) {
    int Nsize = N*size;
    for (int i = 0; i < Nsize; ++i)
        if (A[i] != B[i]) {
            printf("ERROR at %d: %f %f\n", i, A[i], B[i]);
            return false;
        }
    return true;
}
int main() {
    setup();
    int insize = N*C*H*W*sizeof(float);
    int outsize = N*OH*OW*sizeof(float); // total out size
    int ksize = C*S*R*sizeof(float);
    int outh = H - R + 1;  //each out size
    int outw = W - S + 1;
    real* host_input = (real*)malloc(insize); //malloc是按字节分配！大小要乘以字节数！
    real* host_baseline_output = (real*)malloc(outsize);
    real* host_output = (real*)malloc(outsize);
    real* cpu_output = (real*)malloc(outsize);
    real* host_k = (real*)malloc(ksize);
    //init input matrix && output matrix
    for (int n = 0; n < N; ++n) {
        //n=0,1
        for (int c = 0; c < C; ++c) {
            for (int i = 0; i < H; ++i) {
                for (int j = 0; j < W; ++j) {
                    host_input[i*W+j + n*C*H*W + c*H*W] = j;
                }
            }
        }
        for (int i = 0; i < outh; ++i) {
            for (int j = 0; j < outw; ++j) {
                host_output[i*outw+j + n*outh*outw] = 0;
                host_baseline_output[i*outw+j + n*outh*outw] = 0;
            }
        }
    }
    //init kernel(neuron)
    //k=1, c=0,1
    for (int c = 0; c < C; ++c) {
        for (int i = 0; i < R; ++i) {
            for (int j = 0; j < S; ++j) {
                host_k[i*S+j + c*R*S] = i*S+j+1;
            }
        }
    }
    cpu_comp(host_input, cpu_output, host_k, H, W, R, S, outh, outw);
    printf("start...\n");
    printf("---------------------\n");
    //GPU malloc
    real *dev_input, *dev_output, *dev_k;
    hipMalloc(&dev_input, insize);
    hipMalloc(&dev_output, outsize);
    hipMalloc(&dev_k, ksize);
    hipMemcpy(dev_input, host_input, insize, hipMemcpyHostToDevice);
    hipMemcpy(dev_k, host_k, ksize, hipMemcpyHostToDevice);
    hipMemcpy(dev_output, host_output, outsize, hipMemcpyHostToDevice);
    dim3 threadPerBlock(BX, BY);
    dim3 blockPerGrid((outw+BX-1)/BX, (outh+BY-1)/BY);
    //init
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float time;
    //baseline
    hipEventRecord(start, 0);
    baseline<<<blockPerGrid, threadPerBlock>>>(dev_input, dev_output, dev_k, outh, outw);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("baseline: time = %fms \n", time);
    hipMemcpy(host_baseline_output, dev_output, outsize, hipMemcpyDeviceToHost);
    if (check(cpu_output, host_baseline_output, outh*outw)) 
        printf("baseline correct!\n");
    else 
        printf("baseline error!\n");
    printf("---------------------\n");

    //opt_register
    /*hipEventRecord(start, 0);*/
    /*opt_register<<<blockPerGrid, threadPerBlock>>>(dev_input, dev_output, dev_k, outh, outw);*/
    /*hipEventRecord(stop, 0);*/
    /*hipEventSynchronize(stop);*/
    /*hipEventElapsedTime(&time, start, stop);*/
    /*printf("opt_register: time = %fms \n", time);*/
    /*hipMemcpy(host_output, dev_output, outsize, hipMemcpyDeviceToHost);*/
    /*check(cpu_output, host_output, outh*outw);*/
    /*printf("---------------------\n");*/

    //cudnn
    hipEventRecord(start, 0);
    checkCUDNNError(hipdnnConvolutionForward(cudnnHandle, &alpha, bottom_desc_,
                dev_input, filter_desc_, dev_k,
                conv_desc_, algo_, workspace, workspaceSizeInBytes, &beta,
                top_desc_, dev_output));
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("cudnn: time = %fms \n", time);
    hipMemcpy(host_output, dev_output, outsize, hipMemcpyDeviceToHost);

    if (check(cpu_output, host_output, outh*outw)) 
        printf("cudnn correct!\n");
    else
        printf("cudnn error!\n");

    /*printf("cudnn result:\n");*/
    /*for (int n = 0; n < N; ++n) {*/
        /*for (int i = 0; i < OH; ++i) {*/
            /*for (int j = 0; j < OW; ++j) {*/
                /*printf("%f ", host_output[i*OW+j + n*OH*OW]);*/
            /*}*/
            /*printf("\n");*/
        /*}*/
        /*printf(">>>\n");*/
    /*}*/
    /*printf("---------------------\n");*/
    /*printf("cpu result:\n");*/
    /*for (int n = 0; n < N; ++n) {*/
        /*for (int i = 0; i < outh; ++i) {*/
            /*for (int j = 0; j < outw; ++j) {*/
                /*printf("%f ", cpu_output[i*outw+j + n*outh*outw]);*/
            /*}*/
            /*printf("\n");*/
        /*}*/
        /*printf(">>>\n");*/
    /*}*/
    /*printf("\n");*/

    return 0;
}
